/*
cuBLAS global variable definitions
*/
#include "cuda_common.h"
#include "cublas_common.h"
#include <hipblas.h>
#include <hipblaslt.h>

// Define the global variables declared as extern in cublas_common.h
const size_t cublaslt_workspace_size = 32 * 1024 * 1024;
void* cublaslt_workspace = NULL;
hipblasComputeType_t cublas_compute = HIPBLAS_COMPUTE_32F;
hipblasHandle_t cublas_handle;
hipblasLtHandle_t cublaslt_handle;



extern "C" {

    void cublas_init() {
        int deviceIdx = 0;
        cudaCheck(hipSetDevice(deviceIdx));
        hipGetDeviceProperties(&deviceProp, deviceIdx);
        printf("[cublas_init] Device %d: %s\n", deviceIdx, deviceProp.name);
        
    
        // setup cuBLAS and cuBLASLt
        cublasCheck(hipblasCreate(&cublas_handle));
        cublasCheck(hipblasLtCreate(&cublaslt_handle));
        // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
        int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
        printf("enable_tf32: %d\n", enable_tf32);
        cublas_compute = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
        hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
        cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
        // setup the (global) cuBLASLt workspace
        cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));
        fflush(stdout);
    }

}