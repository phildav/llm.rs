#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(const float *a, const float *b, float *out, size_t n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < n) {
        out[idx] = a[idx] + b[idx];
    }
}

extern "C" void add_cuda(const float *a, const float *b, float *out, size_t n) {
    
    add<<<(n+1023)/1024, 1024>>>(a, b, out, n);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error after kernel launch: %s\n", hipGetErrorString(err));
        return;
    }

    hipDeviceSynchronize();
}